#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/device_free.h>
#include <ctime>
#include <fstream>
#include <climits>
#include <cmath>
using namespace std;

struct Point {
    int x, y;
    double demand;

    //adding the parameter for the time window
    double earlyTime;
    double latestTime;   // earliest time to start service
    double serviceTime;  // service time required
  Point() {}
};

int read(string filename, int *h_x, int *h_y, double *h_demand, double *h_earlyTime, double *h_latestTime, double *h_serviceTime) {
    ifstream infile(filename);
    if (!infile) {
        cerr << "Error opening file: " << filename << endl;
        return -1;
    }
    string line;
    for(int i=0;i<4;i++){
        getline(infile,line);
    }
    int vechiceleCapacity,nvechicles;
    infile >>nvechicles>> vechiceleCapacity;

    for(int i=0;i<4;i++){
        getline(infile,line);
    }

    int idx=0;
    while (getline(infile, line)) {
        if (line.empty()) continue; // Skip empty lines
        int no,x, y;
        double demand, earlyTime, latestTime, serviceTime;
        if (!(infile >> no >> x >> y >> demand >> earlyTime >> latestTime >> serviceTime)) {
            //cerr << "Error reading line: " << line << endl;
            continue; // Skip lines that don't match the expected format
        }
        h_x[idx]=x;
        h_y[idx]=y;
        h_demand[idx]=demand;
        h_earlyTime[idx]=earlyTime;
        h_latestTime[idx]=latestTime;
        h_serviceTime[idx]=serviceTime;
        idx++;
    }

    infile.close();
    return idx;
}

__global__ void weightUpdate(int *d_x, int *d_y, float *d_weights, bool *d_inMST, int *d_parent, int current, int nodes) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id==current){
        d_weights[id]=INT_MAX;
        return;
    }
    if(id >= nodes)
        return;
    // if(id==nodes-1){
    //     printf("Current: %d\n",current);
    //     printf("Weights[%d]: %d\n",id,d_weights[id]);
    //     printf("Parent[%d]: %d\n",id,d_parent[id]);
    //     printf("d_x[%d]: %d, d_y[%d]: %d\n",id,d_x[id],id,d_y[id]);
    // }

    if(d_inMST[id])
        return;
    int dx = d_x[current] - d_x[id];
    int dy = d_y[current] - d_y[id];
    float distance = sqrtf(dx * dx + dy * dy); // Squared distance to avoid sqrt for efficiency
    if(!d_inMST[id] && d_weights[id] > distance) {
        d_weights[id] = distance;
        d_parent[id] = current;
    }
    // if(id==20){
    //     printf("Current: %d, Weights[id]: %d\n",current,d_weights[id]);
    // }
}

int main(int argc, char *argv[]){

    // if(argc<1) {
    //     std::cout<<"Please provide the input file name"<<std::endl;
    //     return -1;
    // }
    string filename=argv[1];
    int *h_x,*h_y;
    double *h_demand,*h_earlyTime,*h_latestTime,*h_serviceTime;


    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error: Could not open file " << filename << "\n";
        return 1;
    }

    string line;
    int count = 0;
    while (getline(file, line)) {
        count++;
    }
    h_x=new int[count];
    h_y=new int[count];
    h_demand=new double[count];
    h_earlyTime=new double[count];
    h_latestTime=new double[count];
    h_serviceTime=new double[count];
    //cout<<"Total number of lines in the file: "<<count<<endl;

    int nodes=read(filename,h_x,h_y,h_demand,h_earlyTime,h_latestTime,h_serviceTime);
    // cout<<"Total number of nodes including depot: "<<nodes<<endl;
    // for(int i=0;i<5;i++){
    //     cout<<"Customer "<<i+1<<": ("<<h_x[i]<<", "<<h_y[i]<<"), Demand: "<<h_demand[i]<<", Time Window: ["<<h_earlyTime[i]<<", "<<h_latestTime[i]<<"], Service Time: "<<h_serviceTime[i]<<endl;
    // }
    file.close();
    // Code for MST.
    int *d_x,*d_y;
    double *d_demand,*d_earlyTime,*d_latestTime,*d_serviceTime;
    hipMalloc((void**)&d_x,nodes*sizeof(int));
    hipMalloc((void**)&d_y,nodes*sizeof(int));
    hipMalloc((void**)&d_demand,nodes*sizeof(double));
    hipMalloc((void**)&d_earlyTime,nodes*sizeof(double));
    hipMalloc((void**)&d_latestTime,nodes*sizeof(double));
    hipMalloc((void**)&d_serviceTime,nodes*sizeof(double));
    hipMemcpy(d_x, h_x, nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_demand, h_demand, nodes * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_earlyTime, h_earlyTime, nodes * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_latestTime, h_latestTime, nodes * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_serviceTime, h_serviceTime, nodes * sizeof(double), hipMemcpyHostToDevice);
    
    long long int edge_sum=0;
    int current=0;
    int cnt=0;

    // int *parent=new int[nodes];
    // parent[0]=-1;
    bool *inMST=new bool[nodes];
    vector<float> weights(nodes);
    for(int i=0;i<nodes;i++){
        weights[i]=INT_MAX;
        inMST[i]=false;
    }
    
    weights[0]=0.0f;

    thrust::device_vector<float> d_weights(weights.begin(), weights.end());
    thrust::device_ptr<float> ptr=d_weights.data();
    thrust::device_vector<bool> inMST_d(inMST,inMST+nodes);
    thrust::device_vector<int> d_parent(nodes);
    d_parent[0]=-1;

    // ======================== Main code ====================================
    clock_t begin = clock();

    //cout<<"calling MST kernel"<<endl;
    while(cnt<nodes-1){
        cnt++;
        inMST_d[current]=true;
        weightUpdate<<<1,nodes>>>(d_x, d_y, thrust::raw_pointer_cast(d_weights.data()), thrust::raw_pointer_cast(inMST_d.data()), thrust::raw_pointer_cast(d_parent.data()), current, nodes);
        hipDeviceSynchronize();
        int min_index=thrust::min_element(ptr, ptr + nodes) - ptr;
        //cout<<"Current Node: "<<current<<", Next Node: "<<min_index<<", Weight: "<<d_weights[min_index]<<endl;
        //cout<<"Adding edge cost: "<<d_weights[min_index]<<endl;
        //cout<<"Added edge: "<<min_index<<endl;

        edge_sum+=d_weights[min_index];
        current=min_index;
        d_weights[min_index]=INT_MAX;
    }
    clock_t end = clock();


    for(int i=0;i<nodes;i++){
        cout<<"Node: "<<i<<", Parent: "<<d_parent[i]<<endl;
    }

    // ======================== Results ====================================
    // Print parent of nodes in MST
    // for(int i=0;i<nodes;i++){
    //     if(d_parent[i]!=-1){
    //         cout<<d_parent[i]<<" -- "<<i<<endl;
    //     }
    // }
    cout<<"MST cost: "<<edge_sum<<endl;

    // Print the time for execution
    double elapsed_time = double(end - begin) / CLOCKS_PER_SEC;
    //cout<<"Execution time: "<<elapsed_time<<endl;


    // ======================== Memory Deallocation ====================================
    // thrust::device_free(ptr); 	
    // device_weights.clear();
    // thrust::device_vector<int>().swap(device_weights);
    
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_demand);
    hipFree(d_earlyTime);
    hipFree(d_latestTime);
    hipFree(d_serviceTime);
    delete[] h_x;
    delete[] h_y;
    delete[] h_demand;
    delete[] h_earlyTime;
    delete[] h_latestTime;
    delete[] h_serviceTime;
    delete[] inMST;
    

    return 0;
}

